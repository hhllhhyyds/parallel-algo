#include "hip/hip_runtime.h"
#include "conv_2d_constant_filter.h"
#include "error_assert.h"

#include <hip/hip_runtime.h>

__constant__ float filter[(2 * FILTER_RADIUS + 1) * (2 * FILTER_RADIUS + 1)];

__global__ void
conv_2d_constant_filter_kernel(
    float *in,  // row major 2D matrix
    float *out, // row major 2D matrix
    int width, int height);

__global__ void
conv_2d_tiled_constant_filter_kernel(
    float *in,
    float *out,
    int width, int height);

#ifdef __cplusplus
extern "C"
{
#endif

    void conv_2d_constant_filter_p_dev(float *in,  // row major 2D matrix
                                       float *out, // row major 2D matrix
                                       int width, int height)
    {
        dim3 dim_block;
        dim_block.x = 32;
        dim_block.y = 32;
        dim_block.z = 1;

        dim3 dim_grid;
        dim_grid.x = (width + dim_block.x - 1) / dim_block.x;
        dim_grid.y = (height + dim_block.y - 1) / dim_block.y;
        dim_grid.z = 1;

        conv_2d_constant_filter_kernel<<<dim_grid, dim_block>>>(in, out, width, height);
        cudaCheckErrors("Error in convolution");
    }

    void conv_2d_tiled_constant_filter_p_dev(float *in,  // row major 2D matrix
                                             float *out, // row major 2D matrix
                                             int width, int height)
    {
        dim3 dim_block;
        dim_block.x = IN_TILE_DIM;
        dim_block.y = IN_TILE_DIM;
        dim_block.z = 1;

        dim3 dim_grid;
        dim_grid.x = (width + OUT_TILE_DIM - 1) / OUT_TILE_DIM;
        dim_grid.y = (height + OUT_TILE_DIM - 1) / OUT_TILE_DIM;
        dim_grid.z = 1;

        conv_2d_tiled_constant_filter_kernel<<<dim_grid, dim_block>>>(in, out, width, height);
        cudaCheckErrors("Error in convolution");
    }

    void set_filter_constant(float *filter_h)
    {
        hipMemcpyToSymbol(HIP_SYMBOL(filter), filter_h, (2 * FILTER_RADIUS + 1) * (2 * FILTER_RADIUS + 1) * sizeof(float));
    }

#ifdef __cplusplus
}
#endif

__global__ void
conv_2d_constant_filter_kernel(
    float *in,  // row major 2D matrix
    float *out, // row major 2D matrix
    int width, int height)
{
    const int r = FILTER_RADIUS;
    int out_col = blockIdx.x * blockDim.x + threadIdx.x;
    int out_row = blockIdx.y * blockDim.y + threadIdx.y;

    float val = 0.0f;
    for (int f_row = 0; f_row < 2 * r + 1; ++f_row)
    {
        for (int f_col = 0; f_col < 2 * r + 1; ++f_col)
        {
            int in_row = out_row - r + f_row;
            int in_col = out_col - r + f_col;

            if (in_row >= 0 && in_row <= height && in_col >= 0 && in_col <= width)
            {
                val += filter[f_row * (2 * r + 1) + f_col] * in[in_row * width + in_col];
            }
        }
    }

    out[out_row * width + out_col] = val;
}

__global__ void
conv_2d_tiled_constant_filter_kernel(
    float *in,
    float *out,
    int width, int height)
{
    int col = blockIdx.x * OUT_TILE_DIM + threadIdx.x - FILTER_RADIUS;
    int row = blockIdx.y * OUT_TILE_DIM + threadIdx.y - FILTER_RADIUS;

    __shared__ float tile_in[IN_TILE_DIM][IN_TILE_DIM];
    tile_in[threadIdx.y][threadIdx.x] = (row >= 0 && col >= 0 && row < height && col < width) ? in[row * width + col] : 0.0f;
    __syncthreads();

    int tile_col = threadIdx.x - FILTER_RADIUS;
    int tile_row = threadIdx.y - FILTER_RADIUS;

    if (row >= 0 && col >= 0 && row < height && col < width)
    {
        if (tile_col >= 0 && tile_col < OUT_TILE_DIM && tile_row >= 0 && tile_row < OUT_TILE_DIM)
        {
            float val = 0.0f;

            for (int f_row = 0; f_row < 2 * FILTER_RADIUS + 1; ++f_row)
            {
                for (int f_col = 0; f_col < 2 * FILTER_RADIUS + 1; ++f_col)
                {
                    val += filter[f_row * (2 * FILTER_RADIUS + 1) + f_col] * tile_in[tile_row + f_row][tile_col + f_col];
                }
            }

            out[row * width + col] = val;
        }
    }
}