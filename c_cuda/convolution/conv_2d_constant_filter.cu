#include "hip/hip_runtime.h"
#include "conv_2d_constant_filter.h"
#include "error_assert.h"

#include <hip/hip_runtime.h>

__constant__ float filter[(2 * FILTER_RADIUS + 1) * (2 * FILTER_RADIUS + 1)];

__global__ void
conv_2d_constant_filter_kernel(
    float *in,  // row major 2D matrix
    float *out, // row major 2D matrix
    int width, int height);

#ifdef __cplusplus
extern "C"
{
#endif

    void conv_2d_constant_filter_p_dev(float *in,  // row major 2D matrix
                                       float *out, // row major 2D matrix
                                       int width, int height)
    {
        dim3 dim_block;
        dim_block.x = 32;
        dim_block.y = 32;
        dim_block.z = 1;

        dim3 dim_grid;
        dim_grid.x = (width + dim_block.x - 1) / dim_block.x;
        dim_grid.y = (height + dim_block.y - 1) / dim_block.y;
        dim_grid.z = 1;

        conv_2d_constant_filter_kernel<<<dim_grid, dim_block>>>(in, out, width, height);
        cudaCheckErrors("Error in convolution");
    }

    void set_filter_constant(float *filter_h)
    {
        hipMemcpyToSymbol(HIP_SYMBOL(filter), filter_h, (2 * FILTER_RADIUS + 1) * (2 * FILTER_RADIUS + 1) * sizeof(float));
    }

#ifdef __cplusplus
}
#endif

__global__ void
conv_2d_constant_filter_kernel(
    float *in,  // row major 2D matrix
    float *out, // row major 2D matrix
    int width, int height)
{
    const int r = FILTER_RADIUS;
    int out_col = blockIdx.x * blockDim.x + threadIdx.x;
    int out_row = blockIdx.y * blockDim.y + threadIdx.y;

    float val = 0.0f;
    for (int f_row = 0; f_row < 2 * r + 1; ++f_row)
    {
        for (int f_col = 0; f_col < 2 * r + 1; ++f_col)
        {
            int in_row = out_row - r + f_row;
            int in_col = out_col - r + f_col;

            if (in_row >= 0 && in_row <= height && in_col >= 0 && in_col <= width)
            {
                val += filter[f_row * (2 * r + 1) + f_col] * in[in_row * width + in_col];
            }
        }
    }

    out[out_row * width + out_col] = val;
}