#include "hip/hip_runtime.h"
#include "conv_2d_constant_filter.h"
#include "error_assert.h"

#include <hip/hip_runtime.h>

__constant__ float filter[(2 * FILTER_RADIUS + 1) * (2 * FILTER_RADIUS + 1)];

__global__ void
conv_2d_constant_filter_kernel(
    float *in,  // row major 2D matrix
    float *out, // row major 2D matrix
    int r, int width, int height);

#ifdef __cplusplus
extern "C"
{
#endif

    void conv_2d_constant_filter_p_dev(float *in,       // row major 2D matrix
                                       float *out,      // row major 2D matrix
                                       float *filter_h, // row major 2D matrix
                                       int r, int width, int height)
    {
        if (r != FILTER_RADIUS)
        {
            fprintf(stderr, "Fatal error: filter radius not match, at %s:%d", __FILE__, __LINE__);
            fprintf(stderr, "*** FAILED - ABORTING\n");
            exit(1);
        }

        dim3 dim_block;
        dim_block.x = 32;
        dim_block.y = 32;
        dim_block.z = 1;

        dim3 dim_grid;
        dim_grid.x = (width + dim_block.x - 1) / dim_block.x;
        dim_grid.y = (height + dim_block.y - 1) / dim_block.y;
        dim_grid.z = 1;

        hipMemcpyToSymbol(HIP_SYMBOL(filter), filter_h, (2 * FILTER_RADIUS + 1) * (2 * FILTER_RADIUS + 1) * sizeof(float));

        conv_2d_constant_filter_kernel<<<dim_grid, dim_block>>>(in, out, r, width, height);
        cudaCheckErrors("Error in convolution");
    }

#ifdef __cplusplus
}
#endif

__global__ void
conv_2d_constant_filter_kernel(
    float *in,  // row major 2D matrix
    float *out, // row major 2D matrix
    int r, int width, int height)
{

    int out_col = blockIdx.x * blockDim.x + threadIdx.x;
    int out_row = blockIdx.y * blockDim.y + threadIdx.y;

    float val = 0.0f;
    for (int f_row = 0; f_row < 2 * r + 1; ++f_row)
    {
        for (int f_col = 0; f_col < 2 * r + 1; ++f_col)
        {
            int in_row = out_row - r + f_row;
            int in_col = out_col - r + f_col;

            if (in_row >= 0 && in_row <= height && in_col >= 0 && in_col <= width)
            {
                val += filter[f_row * (2 * r + 1) + f_col] * in[in_row * width + in_col];
            }
        }
    }

    out[out_row * width + out_col] = val;
}